#include "hip/hip_runtime.h"
#include <stdio.h>
#include <string.h>
#include "timer.h"

//#include "config.h"
#include <getopt.h>
#include <stdlib.h>
#include <stdio.h>


#define block_size_x 32
#define block_size_y 16

#define SEED 1234

using namespace std;

struct image_params {
    int input_h;
    int input_w;
    int image_h;
    int image_w;
    int cpu_h;
    int cpu_w;
    int gpu_h;
    int gpu_w;
    int filter_h;
    int filter_w;
    int ratio;
    float* filter;
    float* input;
    float* output;
};

void convolutionSeq(image_params* params) {
    //for each pixel in the output image

    timer sequentialTime = timer("Sequential");
    int input_height = params->input_h;
    int input_width = params->input_w;
    int image_height = params->cpu_h;
    int image_width = params->cpu_w;
    int filter_height = params->filter_h;
    int filter_width = params->filter_w;
    float* input = params->input;
    float* output = params->output;
    float* filter = params->filter;

    sequentialTime.start();

    for (int y = 0; y < image_height; y++) {
        for (int x = 0; x < image_width; x++) {
            output[y * image_width + x] = 0;
            //for each filter weight
            for (int i = 0; i < filter_height; i++) {
                for (int j = 0; j < filter_width; j++) {
                    params -> output[y * image_width + x] += input[(y + i) * input_width + x + j] * filter[i * filter_width + j];
                }
            }
            params -> output[y * image_width + x] /= 35;
        }
    }
    sequentialTime.stop();
    cout << sequentialTime << ",";
}

__constant__ int d_input_width;
__constant__ int d_input_height;
__constant__ int d_filter_width;
__constant__ int d_filter_height;
__constant__ int d_image_width;
__constant__ int d_image_height;

__global__ void convolution_kernel(float* output, float* input, float *d_filter, int start_y) {
    extern __shared__ int s_filter[];
    int linid = threadIdx.x + threadIdx.y * blockDim.x;
    if (linid < d_filter_height * d_filter_width) {
        s_filter[linid] = d_filter[linid];
    }

    unsigned y = blockIdx.y * blockDim.y + threadIdx.y + start_y;
    unsigned x = blockIdx.x * blockDim.x + threadIdx.x;
    unsigned idx = y * d_input_width + x;
    if (x < d_image_width && y < d_image_height) {
        float res = 0.0;
        for (int i = 0; i < d_filter_height; i++) {
            for (int j = 0; j < d_filter_width; j++) {
                res += input[idx + i * d_input_width + j] * s_filter[i * d_filter_width + j];
            }
        }
        output[y * d_image_width + x] = res / 35.0;
    }
}

void convolutionCUDA(image_params *params) {
    int image_height = params->gpu_h;
    int image_width = params->gpu_w;
    int filter_height = params->filter_h;
    int filter_width = params->filter_w;
    int input_height = image_height + filter_height;
    int input_width = params->input_w;
    float* input = params->input + params->cpu_h*params->input_w;
    float * output = params->output + params->cpu_h * params->image_w;
    float * filter = params->filter;

    float* d_input; float* d_output; float *d_filter;
    hipError_t err;
    timer kernelTime = timer("kernelTime");
    timer memoryTime = timer("memoryTime");

    // memory allocation
    err = hipMalloc((void**)&d_input, input_height * input_width * sizeof(float));
    if (err != hipSuccess) { fprintf(stderr, "Error in hipMalloc d_input: %s\n", hipGetErrorString(err)); }
    err = hipMalloc((void**)&d_output, image_height * image_width * sizeof(float));
    if (err != hipSuccess) { fprintf(stderr, "Error in hipMalloc d_output: %s\n", hipGetErrorString(err)); }
    err = hipMalloc((void**)&d_filter, filter_height * filter_width * sizeof(float));
    if (err != hipSuccess) { fprintf(stderr, "Error in hipMalloc d_output: %s\n", hipGetErrorString(err)); }

    kernelTime.start();
    memoryTime.start();

    err = hipMemset(d_output, 0, image_height * image_width * sizeof(float));
    if (err != hipSuccess) { fprintf(stderr, "Error in hipMemset output: %s\n", hipGetErrorString(err)); }
    hipMemcpyAsync(d_filter, filter, sizeof(float) * filter_height * filter_width, hipMemcpyHostToDevice);
    memoryTime.stop();

    dim3 threads(block_size_x, block_size_y);
    const int nstreams = 2;
    int grid_x = (image_width + threads.x - 1) / threads.x;
    int grid_y = ((image_height + threads.y - 1) / threads.y + nstreams - 1) / nstreams;
    dim3 grid(grid_x, grid_y);
    hipStream_t streams[nstreams];
    
    hipMemcpyToSymbol(HIP_SYMBOL("d_input_width"), &input_width, sizeof(int), 0, hipMemcpyHostToDevice);
    hipMemcpyToSymbol(HIP_SYMBOL("d_input_height"), &input_height, sizeof(int), 0, hipMemcpyHostToDevice);
    hipMemcpyToSymbol(HIP_SYMBOL("d_filter_width"), &filter_width, sizeof(int), 0, hipMemcpyHostToDevice);
    hipMemcpyToSymbol(HIP_SYMBOL("d_filter_height"), &filter_height, sizeof(int), 0, hipMemcpyHostToDevice);
    hipMemcpyToSymbol(HIP_SYMBOL("d_image_width"), &image_width, sizeof(int), 0, hipMemcpyHostToDevice);
    hipMemcpyToSymbol(HIP_SYMBOL("d_image_height"), &image_height, sizeof(int), 0, hipMemcpyHostToDevice);

    int border_height = (filter_height / 2) * 2;
    for (int i = 0; i < nstreams; i++) {
        hipStreamCreate(&streams[i]);
        int start_pos = i * threads.y * grid_y;
        int length = threads.y * grid_y + border_height;
        if (start_pos + length > input_height) length = input_height - start_pos;
        hipMemcpyAsync(&d_input[start_pos * input_width], &input[start_pos * input_width], length * input_width * sizeof(float), hipMemcpyHostToDevice, streams[i]);
        convolution_kernel <<<grid, threads, filter_height * filter_width, streams[i] >>> (d_output, d_input, d_filter, start_pos);
    }

    hipDeviceSynchronize();
    memoryTime.start();
    hipMemcpy(output, d_output, image_height * image_width * sizeof(float), hipMemcpyDeviceToHost);
    memoryTime.stop();
    kernelTime.stop();


    if (hipFree(d_input) != hipSuccess) { fprintf(stderr, "Error in freeing d_input: %s\n", hipGetErrorString(err)); }
    if (hipFree(d_output) != hipSuccess) { fprintf(stderr, "Error in freeing d_output: %s\n", hipGetErrorString(err)); }

    cout << memoryTime << ",";
    cout << kernelTime << ",";
}

static void usage(const char* pname)
{
    printf("Usage: %s [OPTION]...\n"
        "\n"
        "  -n NUM     Height.\n"
        "  -m NUM     Width.\n"
        "  -p NUM     GPU Workload Ratio %.\n"
        "  -k NUM     Filter Size.\n"
        , pname);
    exit(0);
}

void read_parameters_f(struct image_params* p, int argc, char** argv)
{
    /* set defaults */
    p->image_h = 1024;
    p->image_w = 1024;
    p->filter_h = 5;
    p->ratio = 0;   
    int ch;

    while ((ch = getopt(argc, argv, "hH:k:m:M:n:N:p:")) != -1)
    {
        switch (ch) {
        case 'm': case 'M': p->image_w = strtol(optarg, 0, 10); break;
        case 'n': case 'N': p->image_h = strtol(optarg, 0, 10); break;
        case 'k': p->filter_h = strtol(optarg, 0, 10); break;
        case 'p': p->ratio = strtol(optarg, 0, 10); break;
        case 'h': default: usage(argv[0]);
        }
    }
    p->ratio = p->ratio < 0 ? 0 : p->ratio;
    p->ratio = p->ratio > 100 ? 100 : p->ratio;

    /*printf("Parameters:\n"
        "  -n %d # number of rows\n"
        "  -m %d # number of columns\n"
        "  -p %d # GPU work share\n"
        "  -k %d # kernel size\n",
        p->image_h, p->image_w, p->ratio, p->filter_h);
    */
}

int main(int argc, char** argv)
{
    // Input Handling
    struct image_params ip;

    read_parameters_f(&ip,argc,argv);

    size_t image_height = ip.image_h;
    size_t image_width = ip.image_w;
    size_t filter_width = ip.filter_h;
    size_t filter_height = ip.filter_h;
    uint32_t ratio = ip.ratio;


    size_t border_height = ((filter_height / 2) * 2);
    size_t border_width = ((filter_width / 2) * 2);
    size_t input_height = (image_height + border_height);
    size_t input_width = (image_width + border_width);

    int i;

    //allocate arrays and fill them
    float* input = (float*)malloc(input_height * input_width * sizeof(float));
    float* output = (float*)calloc(image_height * image_width, sizeof(float));
    float* filter = (float*)malloc(filter_height * filter_width * sizeof(float));

    for (i = 0; i < input_height * input_width; i++) {
        input[i] = (float)(i % SEED);
    }

    // Gaussian Filter Generation

    //This is specific for a W==H smoothening filteri, where W and H are odd.
    for (i = 0; i < filter_height * filter_width; i++) {
        filter[i] = 1.0;
    }
    for (i = filter_width + 1; i < (filter_height - 1) * filter_width; i++) {
        if (i % filter_width > 0 && i % filter_width < filter_width - 1) filter[i] += 1.0;
    }

    filter[filter_width * filter_height / 2] = 3.0;
    //end initialization

    int lower = (image_height * ratio) / 100;

    ip.ratio = ratio;
    ip.input_h = input_height;
    ip.input_w = input_width;
    ip.image_h = image_height;
    ip.image_w = image_width;
    ip.cpu_h = image_height - lower;
    ip.cpu_w = image_width;
    ip.gpu_h = lower;
    ip.gpu_w = image_width;
    ip.filter_h = filter_height;
    ip.filter_w = filter_width;
    ip.filter = filter;
    ip.input = input;
    ip.output = output;

    timer totalTime = timer("Sequential");
    // Run the mixed model
    totalTime.start();
    convolutionSeq(&ip);
    convolutionCUDA(&ip);
    totalTime.stop();
    cout << totalTime << endl;

    free(filter);
    free(input);
    free(output);

    return 0;
}





