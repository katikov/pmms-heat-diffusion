#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <string.h>
#include <hip/hip_runtime.h>
#include <time.h>

#include "input.h"
#include "output.h"

#ifdef FAST
typedef float real;
#else
typedef double real;
#endif

#define MAX(x, y) (((x) > (y)) ? (x) : (y))
#define MIN(x, y) (((x) < (y)) ? (x) : (y))
#define M_SQRT2    1.41421356237309504880

__constant__ __device__ real c_cdir = M_SQRT2 / (M_SQRT2 + 1.0);
__constant__ __device__ real c_cdiag = (M_SQRT2 + 1.0);

static real* maxdiff_block;
static real* min_block;
static real* max_block;
static real* sum_block;
const int threadBlockSize = 128;
const int blockLimit = 65535;

static void checkCudaCall(hipError_t result) {
    if (result != hipSuccess) {
        printf("cuda error \n");
        exit(1);
    }
}

extern "C" 
void* createDeviceMemory(void* p, size_t length){
    real* res;
    checkCudaCall(hipMalloc((void **) &res, length));
    if(res == NULL){
        printf("hipMalloc error\n");
        exit(1);
    }
    if (p) checkCudaCall(hipMemcpy(res, p, length, hipMemcpyHostToDevice));
    return res;
}

extern "C" 
void getDeviceMemory(void* host, void* device, size_t length){
    checkCudaCall(hipMemcpy(host, device, length, hipMemcpyDeviceToHost));
}

extern "C" 
void freeDeviceMemory(void* p){
    checkCudaCall(hipFree(p));
}

__global__ void cudaCopyKernel(unsigned h, unsigned w,
             real* g)
{
    unsigned i = blockIdx.x * blockDim.x + threadIdx.x;
    if(i < h){
        g[i*w+w-1] = g[i*w+1];
        g[i*w] = g[i*w+w-2];
    }
}

__global__ void cudaReduceKernel(unsigned h, unsigned w, real* data, 
                            real* global_min, real* global_max, real* global_sum) {
    unsigned tid = threadIdx.x;
    unsigned i = blockIdx.x+1;
    unsigned j = blockIdx.y*blockDim.x + threadIdx.x + 1;
    unsigned idx = i*w+j;
    __shared__ real min[threadBlockSize];
    __shared__ real max[threadBlockSize];
    __shared__ real sum[threadBlockSize];
    min[tid] = INFINITY; max[tid] = -INFINITY; sum[tid] = 0;
    
    if(j<w-1){
        min[tid] = data[idx];
        max[tid] = data[idx];
        sum[tid] = data[idx];
    }
    __syncthreads();
    for(unsigned s = blockDim.x/2; s>0; s/=2){
        if(tid<s){
            max[tid] = MAX(max[tid], max[tid+s]);
            min[tid] = MIN(min[tid], min[tid+s]);
            sum[tid] += sum[tid+s];
        }
        __syncthreads();
    }

    // for(unsigned s=1; s<blockDim.x; s*=2){
    //     if(tid%(2*s)==0){
    //         max[tid] = MAX(max[tid], max[tid+s]);
    //         min[tid] = MIN(min[tid], min[tid+s]);
    //         sum[tid] += sum[tid+s];
    //     }
    //     __syncthreads();
    // }
    
    if(tid==0){
        unsigned bid = ((w-2+threadBlockSize-1)/threadBlockSize)*blockIdx.x + blockIdx.y;
        global_min[bid] = min[0];
        global_max[bid] = max[0];
        global_sum[bid] = sum[0];
    }

}

__global__ void cudaMaxdiffKernel(int step, int max_id, real* data){
    unsigned i = blockIdx.x * blockDim.x + threadIdx.x;
    if(i<max_id)
        data[i] = MAX(data[i], data[i+step]);
}

// __global__ void cudaPrecomputeKernel(unsigned w, 
//                         real* src, real* temp_result)
// {
//     //size_t tid = threadIdx.x;
//     unsigned i = blockIdx.x+1;
//     unsigned j = blockIdx.y*blockDim.x + threadIdx.x;
//     unsigned idx = i*w+j;
//     if(j<w){
//         temp_result[idx] = src[idx-w] + src[idx+w];
//     }
// }

__global__ void warm_up_gpu() {
    unsigned int tid = blockIdx.x * blockDim.x + threadIdx.x;
    float ia, ib;
    ia = ib = 0.0f;
    ib += ia + tid;
}

extern "C"
void warmup() {
    warm_up_gpu <<< blockLimit, threadBlockSize >>> ();
}

__global__ void cudaComputeKernel(unsigned w, unsigned h,
                        real* src, real* dst, real* c, real* global_maxdiff)
{
    unsigned tid = threadIdx.x;
    unsigned i = blockIdx.x+1;
    unsigned j = blockIdx.y*blockDim.x + threadIdx.x + 1;
    unsigned idx = i*w+j;
    __shared__ real diff[threadBlockSize];
    diff[tid] = 0.0;

    if(j<w-1){

        real weight = c[idx];
        real restw = 1.0 - weight;

        // dst[idx] = weight * src[idx] +
        //      (temp_result[idx] + src[idx+1] + src[idx-1]) * (restw * c_cdir) +
        //      (temp_result[idx-1] + temp_result[idx+1]) * (restw * c_cdiag);

        dst[idx] = weight * src[idx] +
            (src[idx+w] + src[idx-w] + src[idx+1] + src[idx-1]) * (restw * c_cdir) +

            (src[idx-w-1] + src[idx-w+1] + src[idx+w-1] + src[idx+w+1]) * (restw * c_cdiag);
        
        diff[tid] = fabs(dst[idx] - src[idx]);
    }
    
    __syncthreads();

    for(unsigned s = blockDim.x/2; s>0; s/=2){
        if(tid<s){
            diff[tid] = MAX(diff[tid], diff[tid+s]);
        }
        __syncthreads();
    }

    // for(unsigned s=1; s<blockDim.x; s*=2){
    //     if(tid%(2*s)==0)
    //         diff[tid] = MAX(diff[tid], diff[tid+s]);
    //     __syncthreads();
    // }

    if(tid==0){
        unsigned blocksPerLine = (w-2+threadBlockSize-1)/threadBlockSize;
        global_maxdiff[blockIdx.x * blocksPerLine + blockIdx.y] = diff[0];
    }
    
}

// =====================================================================================
// =====================================================================================
// =====================================================================================
// Taken from NVIDIA White Paper on separable convolutions and adapted
// https://developer.download.nvidia.com/compute/cuda/1.1-Beta/x86_64_website/projects/convolutionSeparable/doc/convolutionSeparable.pdf
/*
 * Copyright 1993-2007 NVIDIA Corporation.  All rights reserved.
 *
 * NOTICE TO USER:
 *
 * This source code is subject to NVIDIA ownership rights under U.S. and
 * international Copyright laws.  Users and possessors of this source code
 * are hereby granted a nonexclusive, royalty-free license to use this code
 * in individual and commercial software.
 *
 * NVIDIA MAKES NO REPRESENTATION ABOUT THE SUITABILITY OF THIS SOURCE
 * CODE FOR ANY PURPOSE.  IT IS PROVIDED "AS IS" WITHOUT EXPRESS OR
 * IMPLIED WARRANTY OF ANY KIND.  NVIDIA DISCLAIMS ALL WARRANTIES WITH
 * REGARD TO THIS SOURCE CODE, INCLUDING ALL IMPLIED WARRANTIES OF
 * MERCHANTABILITY, NONINFRINGEMENT, AND FITNESS FOR A PARTICULAR PURPOSE.
 * IN NO EVENT SHALL NVIDIA BE LIABLE FOR ANY SPECIAL, INDIRECT, INCIDENTAL,
 * OR CONSEQUENTIAL DAMAGES, OR ANY DAMAGES WHATSOEVER RESULTING FROM LOSS
 * OF USE, DATA OR PROFITS,  WHETHER IN AN ACTION OF CONTRACT, NEGLIGENCE
 * OR OTHER TORTIOUS ACTION,  ARISING OUT OF OR IN CONNECTION WITH THE USE
 * OR PERFORMANCE OF THIS SOURCE CODE.
 *
 * U.S. Government End Users.   This source code is a "commercial item" as
 * that term is defined at  48 C.F.R. 2.101 (OCT 1995), consisting  of
 * "commercial computer  software"  and "commercial computer software
 * documentation" as such terms are  used in 48 C.F.R. 12.212 (SEPT 1995)
 * and is provided to the U.S. Government only as a commercial end item.
 * Consistent with 48 C.F.R.12.212 and 48 C.F.R. 227.7202-1 through
 * 227.7202-4 (JUNE 1995), all U.S. Government End Users acquire the
 * source code with only those rights set forth herein.
 *
 * Any use of this source code in individual and commercial software must
 * include, in the user documentation and internal comments to the code,
 * the above Disclaimer and U.S. Government End Users Notice.
 */

////////////////////////////////////////////////////////////////////////////////
// Common host and device functions
////////////////////////////////////////////////////////////////////////////////

//Round a / b to nearest higher integer value
int iDivUp(int a, int b) {
    return (a % b != 0) ? (a / b + 1) : (a / b);
}

//Round a / b to nearest lower integer value
int iDivDown(int a, int b) {
    return a / b;
}

//Align a to nearest higher multiple of b
int iAlignUp(int a, int b) {
    return (a % b != 0) ? (a - a % b + b) : a;
}

//Align a to nearest lower multiple of b
int iAlignDown(int a, int b) {
    return a - a % b;
}

////////////////////////////////////////////////////////////////////////////////
// Loop unrolling templates, needed for best performance
////////////////////////////////////////////////////////////////////////////////
#ifdef FAST
#define REAL_SIZE 1
#else
#define REAL_SIZE 2
#endif
#define KERNEL_RADIUS 1
#define KERNEL_W (2 * KERNEL_RADIUS + 1)
#define IMUL(a, b) __mul24(a, b)
#define ROW_TILE_W 128/REAL_SIZE
#define KERNEL_RADIUS_ALIGNED 16 // Half warps are 16 elemenets big
#define COLUMN_TILE_W 16/REAL_SIZE
#define COLUMN_TILE_H 48/REAL_SIZE

__global__ void cudaRowComputeKernel(
    unsigned dataW,
    unsigned dataH,
    real* d_Result,
    real* d_Data
){
        //Data cache
        __shared__ real data[KERNEL_RADIUS + ROW_TILE_W + KERNEL_RADIUS];

        //Current tile and apron limits, relative to row start
        const int         tileStart = IMUL(blockIdx.x, ROW_TILE_W);
        const int           tileEnd = tileStart + ROW_TILE_W - 1;
        const int        apronStart = tileStart - KERNEL_RADIUS;
        const int          apronEnd = tileEnd + KERNEL_RADIUS;

        //Clamp tile and apron limits by image borders
        const int    tileEndClamped = min(tileEnd, dataW - 1);
        const int apronStartClamped = max(apronStart, 0);
        const int   apronEndClamped = min(apronEnd, dataW - 1);

        //Row start index in d_Data[]
        const int          rowStart = IMUL(blockIdx.y, dataW);

        //Aligned apron start. Assuming dataW and ROW_TILE_W are multiples 
        //of half-warp size, rowStart + apronStartAligned is also a 
        //multiple of half-warp size, thus having proper alignment 
        //for coalesced d_Data[] read.
        const int apronStartAligned = tileStart - KERNEL_RADIUS_ALIGNED;

        const int loadPos = apronStartAligned + threadIdx.x;
        //Set the entire data cache contents
        //Load global memory values, if indices are within the image borders,
        //or initialize with zeroes otherwise
        if (loadPos >= apronStart) {
            const int smemPos = loadPos - apronStart;

            data[smemPos] =
                ((loadPos >= apronStartClamped) && (loadPos <= apronEndClamped)) ?
                d_Data[rowStart + loadPos] : 0;
        }


        //Ensure the completness of the loading stage
        //because results, emitted by each thread depend on the data,
        //loaded by another threads
        __syncthreads();
        const int writePos = tileStart + threadIdx.x;
        //Assuming dataW and ROW_TILE_W are multiples of half-warp size,
        //rowStart + tileStart is also a multiple of half-warp size,
        //thus having proper alignment for coalesced d_Result[] write.
        if (writePos <= tileEndClamped) {
            const int smemPos = writePos - apronStart;
            real sum = 0;
            sum = data[smemPos - 1] * c_cdiag + data[smemPos] * c_cdir + data[smemPos + 1] * c_cdiag;
            d_Result[rowStart + writePos] = sum;
        }
    }

__global__ void cudaColumnComputeKernel(
    int dataW,
    int dataH,
    real* d_Src,
    real* d_Result,
    real* d_Data,
    int smemStride,
    int gmemStride,
    real* cond_self,
    real* cond_neighbors,
    real* maxdiff
) {
    //Data cache
    __shared__ real data[COLUMN_TILE_W * (KERNEL_RADIUS + COLUMN_TILE_H + KERNEL_RADIUS)];

    //Current tile and apron limits, in rows
    const int         tileStart = IMUL(blockIdx.y, COLUMN_TILE_H);
    const int           tileEnd = tileStart + COLUMN_TILE_H - 1;
    const int        apronStart = tileStart - KERNEL_RADIUS;
    const int          apronEnd = tileEnd + KERNEL_RADIUS;

    //Clamp tile and apron limits by image borders
    const int    tileEndClamped = min(tileEnd, dataH - 1);
    const int apronStartClamped = max(apronStart, 0);
    const int   apronEndClamped = min(apronEnd, dataH - 1);

    //Current column index
    const int       columnStart = IMUL(blockIdx.x, COLUMN_TILE_W) + threadIdx.x;

    //Shared and global memory indices for current column
    int smemPos = IMUL(threadIdx.y, COLUMN_TILE_W) + threadIdx.x;
    int gmemPos = IMUL(apronStart + threadIdx.y, dataW) + columnStart;
    //Cycle through the entire data cache
    //Load global memory values, if indices are within the image borders,
    //or initialize with zero otherwise
    for (int y = apronStart + threadIdx.y; y <= apronEnd; y += blockDim.y) {
        data[smemPos] =
            ((y >= apronStartClamped) && (y <= apronEndClamped)) ?
            d_Data[gmemPos] : 0;
        smemPos += smemStride;
        gmemPos += gmemStride;
    }

    //Ensure the completness of the loading stage
    //because results, emitted by each thread depend on the data, 
    //loaded by another threads
    __syncthreads();
    //Shared and global memory indices for current column
    smemPos = IMUL(threadIdx.y + KERNEL_RADIUS, COLUMN_TILE_W) + threadIdx.x;
    gmemPos = IMUL(tileStart + threadIdx.y, dataW) + columnStart;
    //Cycle through the tile body, clamped by image borders
    //Calculate and output the results
    int start = max(tileStart + threadIdx.y, 1);
    int end = min(tileEndClamped, dataH - 2);
    real res;
    real source;
    for (int y = start; y <= end; y += blockDim.y) {
        real sum = 0;
        sum = data[smemPos - 1] * 0.25 + data[smemPos] * (0.25 * c_cdir / c_cdiag) + data[smemPos + 1] * 0.25;
        source = d_Src[gmemPos];
        res = sum * cond_neighbors[gmemPos] + source * cond_self[gmemPos];
        d_Result[gmemPos] = res;
        maxdiff[gmemPos] = abs(source - res);
        smemPos += smemStride;
        gmemPos += gmemStride;
    }
}

// End of Code for separable convolution taken and adapte from NVIDIA White Paper
// =====================================================================================
// =====================================================================================
// =====================================================================================

extern "C"
void cuda_init(size_t h, size_t w){
    size_t blocksPerLine = (w-2+threadBlockSize-1)/threadBlockSize;
    size_t num_blocks = (h-2)*blocksPerLine;
    //checkCudaCall(hipMalloc((void **) &maxdiff_block, h*w * sizeof(double)));
    checkCudaCall(hipMalloc((void **) &maxdiff_block, num_blocks * sizeof(real)));
    checkCudaCall(hipMalloc((void **) &min_block, num_blocks * sizeof(real)));
    checkCudaCall(hipMalloc((void **) &max_block, num_blocks * sizeof(real)));
    checkCudaCall(hipMalloc((void **) &sum_block, num_blocks * sizeof(real)));
}

extern "C"
void cuda_finalize(){
    checkCudaCall(hipFree(maxdiff_block));
    checkCudaCall(hipFree(min_block));
    checkCudaCall(hipFree(max_block));
    checkCudaCall(hipFree(sum_block));
}


extern "C"
double cuda_do_compute_step(size_t h, size_t w, 
                        real* src, real* dst, real* tmp,
                        real* cond_self, real * cond_neighbors){

    dim3 blockGridRows(iDivUp(w, ROW_TILE_W), h);
    dim3 blockGridColumns(iDivUp(w, COLUMN_TILE_W), iDivUp(h, COLUMN_TILE_H));
    dim3 threadBlockRows(KERNEL_RADIUS_ALIGNED + ROW_TILE_W + KERNEL_RADIUS);
    dim3 threadBlockColumns(COLUMN_TILE_W, 8);
    size_t blocksPerLine = (w - 2 + threadBlockSize - 1) / threadBlockSize;
    dim3 numBlocks(h - 2, blocksPerLine);
    dim3 numBlocks2(h - 2, (w + threadBlockSize - 1) / threadBlockSize);

    cudaRowComputeKernel <<<blockGridRows, threadBlockRows >>> (w, h, src, tmp);
    cudaColumnComputeKernel <<<blockGridColumns, threadBlockColumns >>>
        (w, h, 
         src, tmp, dst, 
         COLUMN_TILE_W * threadBlockColumns.y, 
         w * threadBlockColumns.y,
         cond_self, cond_neighbors, maxdiff_block);
    hipMemcpy(&dst[0], &dst[(h - 2) * w], w * sizeof(real), hipMemcpyDeviceToDevice);
    hipMemcpy(&dst[(h - 1)], &dst[1 * w], w * sizeof(real), hipMemcpyDeviceToDevice);

    int len;
    for(len=blocksPerLine*(h-2); len>=threadBlockSize*8; len=(len+1)/2){
        cudaMaxdiffKernel<<<(len+threadBlockSize-1)/threadBlockSize, threadBlockSize>>>((len+1)/2, len/2, maxdiff_block);
    }
    hipDeviceSynchronize();

    
    // real r[blocksPerLine*(h-2)];
    // getDeviceMemory(r, maxdiff_block, blocksPerLine*(h-2)*sizeof(real));
    // real res=0;
    // for(int i=0;i<blocksPerLine*(h-2);i++){
    //     res = MAX(res, r[i]);
    // }

    real* r = (real * )malloc(sizeof(real)*len);
    getDeviceMemory(r, maxdiff_block, len*sizeof(real));
    real res=0;
    for(int i=0;i<len;i++){
        res = MAX(res, r[i]);
    }
    return res;
    
}

extern "C"
void cuda_fill_report(const struct parameters *p, struct results *r,
                        size_t h, size_t w, real* data,
                        struct timespec *before)
{
    // compute min/max/avg 
    real tmin = INFINITY, tmax = -INFINITY;
    real sum = 0.0;
    struct timespec after;

    size_t blocksPerLine = (w-2+threadBlockSize-1)/threadBlockSize;
    dim3 numBlocks(h-2,blocksPerLine);

    //We have said that the final reduction does not need to be included.
    clock_gettime(CLOCK_MONOTONIC, &after);
    
    cudaReduceKernel<<<numBlocks, threadBlockSize>>>(h, w, data, min_block, max_block, sum_block);
    hipDeviceSynchronize();

    real * recv = (real * )malloc(sizeof(real)*blocksPerLine*(h-2));
    getDeviceMemory(recv, min_block, blocksPerLine*(h-2)*sizeof(real));
    for(int i=0;i<blocksPerLine*(h-2);i++){
        tmin = MIN(tmin, recv[i]);
    }

    getDeviceMemory(recv, max_block, blocksPerLine*(h-2)*sizeof(real));
    for(int i=0;i<blocksPerLine*(h-2);i++){
        tmax = MAX(tmax, recv[i]);
    }

    getDeviceMemory(recv, sum_block, blocksPerLine*(h-2)*sizeof(real));
    for(int i=0;i<blocksPerLine*(h-2);i++){
        sum += recv[i];
    }
    r->tmin = tmin;
    r->tmax = tmax;
    r->tavg = sum / (p->N * p->M);
    
    r->time = (real)(after.tv_sec - before->tv_sec) + 
        (real)(after.tv_nsec - before->tv_nsec) / 1e9;
    
}

