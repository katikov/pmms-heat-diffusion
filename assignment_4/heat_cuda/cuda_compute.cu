#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <string.h>
#include <hip/hip_runtime.h>
#include <time.h>

#include "input.h"
#include "output.h"

#define MAX(x, y) (((x) > (y)) ? (x) : (y))
#define MIN(x, y) (((x) < (y)) ? (x) : (y))

__constant__ __device__ double c_cdir = 0.25 * M_SQRT2 / (M_SQRT2 + 1.0);
__constant__ __device__ double c_cdiag = 0.25 / (M_SQRT2 + 1.0);
static double* maxdiff_block;
static double* min_block;
static double* max_block;
static double* sum_block;
const int threadBlockSize = 128;

static void checkCudaCall(hipError_t result) {
    if (result != hipSuccess) {
        printf("cuda error \n");
        exit(1);
    }
}

extern "C" 
void* createDeviceMemory(void* p, size_t length){
    double* res;
    checkCudaCall(hipMalloc((void **) &res, length));
    if(res == NULL){
        printf("hipMalloc error\n");
        exit(1);
    }
    checkCudaCall(hipMemcpy(res, p, length, hipMemcpyHostToDevice));
    return res;
}

extern "C" 
void getDeviceMemory(void* host, void* device, size_t length){
    checkCudaCall(hipMemcpy(host, device, length, hipMemcpyDeviceToHost));
}

extern "C" 
void freeDeviceMemory(void* p){
    checkCudaCall(hipFree(p));
}



__global__ void vectorAddKernel(float* deviceA, float* deviceB, float* deviceResult) {
    unsigned i = blockIdx.x * blockDim.x + threadIdx.x;
// insert operation here
    deviceResult[i] = deviceA[i]+deviceB[i];
}

__global__ void cudaCopyKernel(unsigned h, unsigned w,
             double* g)
{
    unsigned i = blockIdx.x * blockDim.x + threadIdx.x;
    if(i < h){
        g[i*w+w-1] = g[i*w+1];
        g[i*w] = g[i*w+w-2];
    }
}

__global__ void cudaReduceKernel(unsigned h, unsigned w, double* data, 
                            double* global_min, double* global_max, double* global_sum) {
    unsigned tid = threadIdx.x;
    unsigned i = blockIdx.x+1;
    unsigned j = blockIdx.y*blockDim.x + threadIdx.x + 1;
    unsigned idx = i*w+j;
    __shared__ double min[threadBlockSize];
    __shared__ double max[threadBlockSize];
    __shared__ double sum[threadBlockSize];
    min[tid] = INFINITY; max[tid] = -INFINITY; sum[tid] = 0;
    
    if(j<w-1){
        min[tid] = data[idx];
        max[tid] = data[idx];
        sum[tid] = data[idx];
    }
    __syncthreads();
    for(unsigned s = blockDim.x/2; s>0; s/=2){
        if(tid<s){
            max[tid] = MAX(max[tid], max[tid+s]);
            min[tid] = MIN(min[tid], min[tid+s]);
            sum[tid] += sum[tid+s];
        }
        __syncthreads();
    }

    // for(unsigned s=1; s<blockDim.x; s*=2){
    //     if(tid%(2*s)==0){
    //         max[tid] = MAX(max[tid], max[tid+s]);
    //         min[tid] = MIN(min[tid], min[tid+s]);
    //         sum[tid] += sum[tid+s];
    //     }
    //     __syncthreads();
    // }
    
    if(tid==0){
        unsigned bid = ((w-2+threadBlockSize-1)/threadBlockSize)*blockIdx.x + blockIdx.y;
        global_min[bid] = min[0];
        global_max[bid] = max[0];
        global_sum[bid] = sum[0];
    }

}

__global__ void cudaMaxdiffKernel(int step, int max_id, double* data){
    unsigned i = blockIdx.x * blockDim.x + threadIdx.x;
    if(i<max_id)
        data[i] = MAX(data[i], data[i+step]);
}

// __global__ void cudaPrecomputeKernel(unsigned w, 
//                         double* src, double* temp_result)
// {
//     //size_t tid = threadIdx.x;
//     unsigned i = blockIdx.x+1;
//     unsigned j = blockIdx.y*blockDim.x + threadIdx.x;
//     unsigned idx = i*w+j;
//     if(j<w){
//         temp_result[idx] = src[idx-w] + src[idx+w];
//     }
// }


__global__ void cudaComputeKernel(unsigned w, 
                        double* src, double* dst, double* c, double* global_maxdiff)
{
    unsigned tid = threadIdx.x;
    unsigned i = blockIdx.x+1;
    unsigned j = blockIdx.y*blockDim.x + threadIdx.x + 1;
    unsigned idx = i*w+j;
    __shared__ double diff[threadBlockSize];
    diff[tid] = 0.0;

    if(j<w-1){

        double weight = c[idx];
        double restw = 1.0 - weight;

        // dst[idx] = weight * src[idx] +
        //      (temp_result[idx] + src[idx+1] + src[idx-1]) * (restw * c_cdir) +
        //      (temp_result[idx-1] + temp_result[idx+1]) * (restw * c_cdiag);

        dst[idx] = weight * src[idx] +
            (src[idx+w] + src[idx-w] + src[idx+1] + src[idx-1]) * (restw * c_cdir) +

            (src[idx-w-1] + src[idx-w+1] + src[idx+w-1] + src[idx+w+1]) * (restw * c_cdiag);
        
        diff[tid] = fabs(dst[idx] - src[idx]);
    }
    
    __syncthreads();

    for(unsigned s = blockDim.x/2; s>0; s/=2){
        if(tid<s){
            diff[tid] = MAX(diff[tid], diff[tid+s]);
        }
        __syncthreads();
    }

    // for(unsigned s=1; s<blockDim.x; s*=2){
    //     if(tid%(2*s)==0)
    //         diff[tid] = MAX(diff[tid], diff[tid+s]);
    //     __syncthreads();
    // }

    if(tid==0){
        unsigned blocksPerLine = (w-2+threadBlockSize-1)/threadBlockSize;
        global_maxdiff[blockIdx.x * blocksPerLine + blockIdx.y] = diff[0];
    }
    
}

extern "C"
void cuda_init(size_t h, size_t w){
    size_t blocksPerLine = (w-2+threadBlockSize-1)/threadBlockSize;
    size_t num_blocks = (h-2)*blocksPerLine;
    //checkCudaCall(hipMalloc((void **) &maxdiff_block, h*w * sizeof(double)));
    checkCudaCall(hipMalloc((void **) &maxdiff_block, num_blocks * sizeof(double)));
    checkCudaCall(hipMalloc((void **) &min_block, num_blocks * sizeof(double)));
    checkCudaCall(hipMalloc((void **) &max_block, num_blocks * sizeof(double)));
    checkCudaCall(hipMalloc((void **) &sum_block, num_blocks * sizeof(double)));
}

extern "C"
void cuda_finalize(){
    checkCudaCall(hipFree(maxdiff_block));
    checkCudaCall(hipFree(min_block));
    checkCudaCall(hipFree(max_block));
    checkCudaCall(hipFree(sum_block));
}

extern "C"
double cuda_do_compute_step(size_t h, size_t w, 
                        double* src, double* dst, double* c){

        
    cudaCopyKernel<<<(h+threadBlockSize-1)/threadBlockSize, threadBlockSize>>>(h, w, src);
    size_t blocksPerLine = (w-2+threadBlockSize-1)/threadBlockSize;
    dim3 numBlocks(h-2,blocksPerLine);
    dim3 numBlocks2(h-2, (w+threadBlockSize-1)/threadBlockSize);

    ///cudaPrecomputeKernel<<<numBlocks2 ,threadBlockSize>>>(w, src, temp_sum);

    
    cudaComputeKernel<<<numBlocks, threadBlockSize>>>(w, src, dst, c, maxdiff_block);  
    

    int len;
    for(len=blocksPerLine*(h-2); len>=threadBlockSize*8; len=(len+1)/2){
        cudaMaxdiffKernel<<<(len+threadBlockSize-1)/threadBlockSize, threadBlockSize>>>((len+1)/2, len/2, maxdiff_block);
    }
    hipDeviceSynchronize();

    
    // double r[blocksPerLine*(h-2)];
    // getDeviceMemory(r, maxdiff_block, blocksPerLine*(h-2)*sizeof(double));
    // double res=0;
    // for(int i=0;i<blocksPerLine*(h-2);i++){
    //     res = MAX(res, r[i]);
    // }

    double r[len];
    getDeviceMemory(r, maxdiff_block, len*sizeof(double));
    double res=0;
    for(int i=0;i<len;i++){
        res = MAX(res, r[i]);
    }
    return res;
    
}

extern "C"
void cuda_fill_report(const struct parameters *p, struct results *r,
                        size_t h, size_t w, double* data,
                        struct timespec *before)
{
    // compute min/max/avg 
    double tmin = INFINITY, tmax = -INFINITY;
    double sum = 0.0;
    struct timespec after;

    size_t blocksPerLine = (w-2+threadBlockSize-1)/threadBlockSize;
    dim3 numBlocks(h-2,blocksPerLine);

    //We have said that the final reduction does not need to be included.
    clock_gettime(CLOCK_MONOTONIC, &after);
    
    cudaReduceKernel<<<numBlocks, threadBlockSize>>>(h, w, data, min_block, max_block, sum_block);
    hipDeviceSynchronize();

    double recv[blocksPerLine*(h-2)];
    getDeviceMemory(recv, min_block, blocksPerLine*(h-2)*sizeof(double));
    for(int i=0;i<blocksPerLine*(h-2);i++){
        tmin = MIN(tmin, recv[i]);
    }

    getDeviceMemory(recv, max_block, blocksPerLine*(h-2)*sizeof(double));
    for(int i=0;i<blocksPerLine*(h-2);i++){
        tmax = MAX(tmax, recv[i]);
    }

    getDeviceMemory(recv, sum_block, blocksPerLine*(h-2)*sizeof(double));
    for(int i=0;i<blocksPerLine*(h-2);i++){
        sum += recv[i];
    }
    r->tmin = tmin;
    r->tmax = tmax;
    r->tavg = sum / (p->N * p->M);
    
    r->time = (double)(after.tv_sec - before->tv_sec) + 
        (double)(after.tv_nsec - before->tv_nsec) / 1e9;
    
}

